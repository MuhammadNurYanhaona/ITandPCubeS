#include "lpu_parts_tracking.h"
#include "../utils/list.h"
#include "../gpu-utils/gpu_utils.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdlib>

/************************************************************************************************************************************************* 
                             Note that *_cuda.cu files have the code that needs the NVCC compiler for compilation
*************************************************************************************************************************************************/

//----------------------------------------------------------- Property Buffer Manager -----------------------------------------------------------/

void PropertyBufferManager::prepareGpuBuffers() {
	
	check_error(hipMalloc((void **) &gpuBuffer, bufferSize), *logFile);
	check_error(hipMemcpyAsync(gpuBuffer, cpuBuffer, bufferSize, hipMemcpyHostToDevice, 0), *logFile);
	
	check_error(hipMalloc((void **) &gpuPartIndexBuffer, bufferReferenceCount * sizeof(int)), *logFile);
	check_error(hipMemcpyAsync(gpuPartIndexBuffer, cpuPartIndexBuffer, 
			bufferReferenceCount * sizeof(int), hipMemcpyHostToDevice, 0), *logFile);
	
	check_error(hipMalloc((void **) &gpuPartRangeBuffer, partRangeBufferSize * sizeof(int)), *logFile);
	check_error(hipMemcpyAsync(gpuPartRangeBuffer, cpuPartRangeBuffer, 
			partRangeBufferSize * sizeof(int), hipMemcpyHostToDevice, 0), *logFile);
	
	check_error(hipMalloc((void **) &gpuPartBeginningBuffer, 
			bufferEntryCount * sizeof(long int)), *logFile);
	check_error(hipMemcpyAsync(gpuPartBeginningBuffer, cpuPartBeginningBuffer, 
			bufferEntryCount * sizeof(long int), hipMemcpyHostToDevice, 0), *logFile);
}

void PropertyBufferManager::syncDataPartsFromBuffer(List<LpuDataPart*> *dataPartsList) {
	
	check_error(hipMemcpy(cpuBuffer, gpuBuffer, bufferSize, hipMemcpyDeviceToHost), *logFile);

	int currentIndex = 0;
	for (int i = 0; i < dataPartsList->NumElements(); i++) {
		LpuDataPart *dataPart = dataPartsList->Nth(i);
		char *dataStart = cpuBuffer + currentIndex;
		void *data = dataPart->getData();
		int size = dataPart->getSize();
		memcpy(data, dataStart, size);
		currentIndex += size;
	}
}

void PropertyBufferManager::cleanupBuffers() {
	
	bufferSize = 0;
	bufferEntryCount = 0;
	bufferReferenceCount = 0;
	partRangeDepth = 0;
	
	free(cpuBuffer);
	cpuBuffer = NULL;
	free(cpuPartIndexBuffer); 
        cpuPartIndexBuffer = NULL;
	free(cpuPartRangeBuffer); 
        cpuPartRangeBuffer = NULL;
	free(cpuPartBeginningBuffer);
        cpuPartBeginningBuffer = NULL;
       
	hipFree(gpuBuffer); 
	gpuBuffer = NULL;
	hipFree(gpuPartIndexBuffer);
        gpuPartIndexBuffer = NULL;
	hipFree(gpuPartRangeBuffer);
        gpuPartRangeBuffer = NULL;
	hipFree(gpuPartBeginningBuffer);
        gpuPartBeginningBuffer = NULL;
}

//------------------------------------------------------- Versioned Property Buffer Manager -----------------------------------------------------/

void VersionedPropertyBufferManager::prepareGpuBuffers() {
	PropertyBufferManager::prepareGpuBuffers();
	check_error(hipMalloc((void **) &gpuDataPartVersions, bufferEntryCount * sizeof(short)), *logFile);
	check_error(hipMemcpyAsync(gpuDataPartVersions, cpuDataPartVersions, 
			bufferEntryCount * sizeof(short), hipMemcpyHostToDevice, 0), *logFile);
}

void VersionedPropertyBufferManager::syncDataPartsFromBuffer(List<LpuDataPart*> *dataPartsList) {

	check_error(hipMemcpyAsync(cpuBuffer, gpuBuffer, bufferSize, hipMemcpyDeviceToHost, 0), *logFile);
	check_error(hipMemcpy(cpuDataPartVersions, gpuDataPartVersions, 
			bufferEntryCount * sizeof(short), hipMemcpyDeviceToHost), *logFile);

	int currentIndex = 0;
	for (int i = 0; i < bufferEntryCount; i++) {
		VersionedLpuDataPart *dataPart = (VersionedLpuDataPart *) dataPartsList->Nth(i);
		short currVersion = cpuDataPartVersions[i];
		int sizePerVersion = dataPart->getSize() / versionCount;
		for (int j = 0; j < versionCount; j++) {
			int versionIndex = (currVersion + j) % versionCount;
			char *dataStart = cpuBuffer + currentIndex + versionIndex * sizePerVersion;
			void *data = dataPart->getDataVersion(versionIndex);
			memcpy(data, dataStart, sizePerVersion);
		}
		currentIndex += dataPart->getSize();
	}
}

void VersionedPropertyBufferManager::cleanupBuffers() {
	
	PropertyBufferManager::cleanupBuffers();

	free(cpuDataPartVersions);
        cpuDataPartVersions = NULL;       
	hipFree(gpuDataPartVersions); 
	gpuDataPartVersions = NULL;
}
