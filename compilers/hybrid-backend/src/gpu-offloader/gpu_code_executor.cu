#include "hip/hip_runtime.h"
#include "gpu_code_executor.h"
#include "lpu_parts_tracking.h"
#include "../runtime/structure.h"
#include "../gpu-utils/gpu_utils.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <vector>

//-------------------------------------------------------- Offload Statistics -------------------------------------------------------------/

OffloadStats::OffloadStats() {
	timeSpentStagingIn = 0;
	timeSpentExecution = 0;
	timeSpentStagingOut = 0;
}

void OffloadStats::describe(std::ofstream &logFile) {
	logFile << "Overall time spent staging data into GPU from CPU: ";
	logFile << timeSpentStagingIn << " Seconds\n";
	logFile << "Overall time spent executing kernels for the LPUs: ";
	logFile << timeSpentExecution << " Seconds\n";
	logFile << "Overall time spent staging data out from GPU to CPU: ";
	logFile << timeSpentStagingOut << " Seconds\n";
}

//--------------------------------------------------------- GPU Code Executor -------------------------------------------------------------/

GpuCodeExecutor::GpuCodeExecutor(LpuBatchController *lpuBatchController) {
	this->lpuBatchController = lpuBatchController;
}

void GpuCodeExecutor::submitNextLpu(LPU *lpu) {

	if (lpuBatchController->canAddNewLpu() && lpuBatchController->canHoldLpu(lpu)) {
		if (lpuBatchController->isEmptyBatch()) {
			currentBatchLpuRange = Range(lpu->id);
		} else {
			currentBatchLpuRange.max++;
		}
		lpuBatchController->addLpuToTheCurrentBatch(lpu);
		return;	
	}
	if (!lpuBatchController->isEmptyBatch()) {
		execute();
	}

	lpuBatchController->addLpuToTheCurrentBatch(lpu);
	currentBatchLpuRange = Range(lpu->id);
}

void GpuCodeExecutor::forceExecution() {
	if (!lpuBatchController->isEmptyBatch()) {
		execute();
	}
}

void GpuCodeExecutor::submitNextLpus(std::vector<LPU*> *lpuVector) {
	for (unsigned int i = 0; i < lpuVector->size(); i++) {
		LPU *lpu = lpuVector->at(i);
		if (lpu != NULL) {
			submitNextLpu(lpu);
		}
	}
}

void GpuCodeExecutor::execute() {
	
	struct timeval tv;
        gettimeofday(&tv, NULL);
        long startTime = tv.tv_sec * 1000000 + tv.tv_usec;

	lpuBatchController->submitCurrentBatchToGpu();

	gettimeofday(&tv, NULL);
        long endTime = tv.tv_sec * 1000000 + tv.tv_usec;
	double timeTaken = ((endTime - startTime) * 1.0) / (1000 * 1000);
	offloadStats->addStagingInTime(timeTaken);

	startTime = endTime;
	
	offloadFunction();
	hipDeviceSynchronize();
	check_error(hipGetLastError(), *logFile);
	
	gettimeofday(&tv, NULL);
        endTime = tv.tv_sec * 1000000 + tv.tv_usec;
	timeTaken = ((endTime - startTime) * 1.0) / (1000 * 1000);
	offloadStats->addExecutionTime(timeTaken);

	startTime = endTime;

	lpuBatchController->updateBatchDataPartsFromGpuResults();
	lpuBatchController->resetController();

	gettimeofday(&tv, NULL);
        endTime = tv.tv_sec * 1000000 + tv.tv_usec;
	timeTaken = ((endTime - startTime) * 1.0) / (1000 * 1000);
	offloadStats->addStagingOutTime(timeTaken);
}

void GpuCodeExecutor::initialize() {
	this->offloadStats = new OffloadStats();
}

void GpuCodeExecutor::cleanup() { 
	hipDeviceReset(); 
	offloadStats->describe(*logFile);
}
