#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "mmm_gpu_execution.h"
#include "../../test-case/mmm_structure.h"
#include "../../runtime/structure.h"
#include "../../gpu-utils/gpu_constant.h"
#include "../../gpu-offloader/gpu_code_executor.h"
#include "../../gpu-offloader/lpu_parts_tracking.h"
#include "../../utils/list.h"
#include "../../gpu-utils/gpu_partition.h"
#include "../../gpu-utils/gpu_utils.h"

//----------------------------------------------------- MMM Batch LPU Controller -----------------------------------------------------/

MMMLpuBatchController::MMMLpuBatchController(int lpuCountThreshold, long memLimit)  : LpuBatchController() {
	
	List<const char*> *propertyNames = new List<const char*>;
	propertyNames->Append("a");
	propertyNames->Append("b");
	propertyNames->Append("c");
	
	List<const char*> *toBeModifiedProperties = new List<const char*>;
	toBeModifiedProperties->Append("c");

	initialize(lpuCountThreshold, memLimit, propertyNames, toBeModifiedProperties);		
}

int MMMLpuBatchController::calculateLpuMemoryRequirement(LPU *lpu) {
	MMMLpu *mmmLpu = (MMMLpu *) lpu;
	int size = 0;
	if (!dataPartTracker->isAlreadyIncluded(mmmLpu->aPartId, "a")) {
		size += (mmmLpu->aPartDims[0].storage.getLength() 
			* mmmLpu->aPartDims[1].storage.getLength()) * sizeof(double);
	}
	if (!dataPartTracker->isAlreadyIncluded(mmmLpu->bPartId, "b")) {
		size += (mmmLpu->bPartDims[0].storage.getLength() 
			* mmmLpu->bPartDims[1].storage.getLength()) * sizeof(double);
	}
	if (!dataPartTracker->isAlreadyIncluded(mmmLpu->aPartId, "c")) {
		size += (mmmLpu->cPartDims[0].storage.getLength() 
			* mmmLpu->cPartDims[1].storage.getLength()) * sizeof(double);
	}
	return size;
}

void MMMLpuBatchController::addLpuToTheCurrentBatch(LPU *lpu) {
	
	MMMLpu *mmmLpu = (MMMLpu *) lpu;

	LpuDataPart *aPart = new LpuDataPart(2, 
			mmmLpu->aPartDims, mmmLpu->a, sizeof(double), mmmLpu->aPartId);
	bool notRedundant = dataPartTracker->addDataPart(aPart, "a");
	if (!notRedundant) {
		delete aPart;
	}
	LpuDataPart *bPart = new LpuDataPart(2, 
			mmmLpu->bPartDims, mmmLpu->b, sizeof(double), mmmLpu->bPartId);
	notRedundant = dataPartTracker->addDataPart(bPart, "b");
	if (!notRedundant) {
		delete bPart;
	}
	LpuDataPart *cPart = new LpuDataPart(2, 
			mmmLpu->cPartDims, mmmLpu->c, sizeof(double), mmmLpu->cPartId);
	notRedundant = dataPartTracker->addDataPart(cPart, "c");
	if (!notRedundant) {
		delete cPart;
	}

	LpuBatchController::addLpuToTheCurrentBatch(lpu);
}

//------------------------------------------------------ Offloading GPU Kernels ------------------------------------------------------/

__global__ void matrixMultiplyKernel(MMMLpuBatchRange batchRange, 
		mmm::Partition partition, 
		mmm::ArrayMetadata arrayMetadata,
		mmm::TaskGlobals *taskGlobals,
		mmm::ThreadLocals *threadLocals,	
		GpuBufferReferences aBuffers, 
		GpuBufferReferences bBuffers, 
		GpuBufferReferences cBuffers) {

	/*----------------------------------------------------------------------------------------------------------------------------
						    Space A: Top-most User Defined Space 
	----------------------------------------------------------------------------------------------------------------------------*/

	// before we can do anything in the kernel, we need to determine the thread, warp, and sm IDs of the thread
	// executing the kernel code
        int smId = blockIdx.x;
        int warpId = threadIdx.x / WARP_SIZE;
	int threadId = threadIdx.x % WARP_SIZE;
	
	// variables for holding the data part references for the top-space LPU
	double *a, *b, *c;

	// variables for tracking storage and partition dimensions of the top space LPU's data parts
	__shared__ int aSRanges[2][2], bSRanges[2][2], cSRanges[2][2];
	__shared__ int aPRanges[2][2], bPRanges[2][2], cPRanges[2][2];

	// SMs stride over different indexes to get different LPUs to operate on
	Range lpuIdRange = batchRange.lpuIdRange;
	for (int linearId = lpuIdRange.min + smId; linearId <= lpuIdRange.max; linearId += BLOCK_COUNT) {

		// point the a, b, c matrix references to the memory addresses where corresponding data parts for the
		// current LPUs starts 

		if (warpId == 0 && threadId == 0) {
			//------------------------------------------------------------- retrieve a and its dimensions
			int lpuIndex = linearId - lpuIdRange.min;
			int aIndex = aBuffers.partIndexBuffer[lpuIndex];
			int aStartsAt = aBuffers.partBeginningBuffer[aIndex];
			a = (double *) aBuffers.dataBuffer[aStartsAt];
			int aDimRangeStart = lpuIndex * 2 * 2 * 2; // there are storage and partition ranges each has
								   // two integers and the data structure is 2D
			aSRanges[0][0] = aBuffers.partBeginningBuffer[aDimRangeStart];
			aSRanges[0][1] = aBuffers.partBeginningBuffer[aDimRangeStart + 1];
			aSRanges[1][0] = aBuffers.partBeginningBuffer[aDimRangeStart + 2];
			aSRanges[1][1] = aBuffers.partBeginningBuffer[aDimRangeStart + 3];

			aPRanges[0][0] = aBuffers.partBeginningBuffer[aDimRangeStart + 4];
			aPRanges[0][1] = aBuffers.partBeginningBuffer[aDimRangeStart + 5];
			aPRanges[1][0] = aBuffers.partBeginningBuffer[aDimRangeStart + 6];
			aPRanges[1][1] = aBuffers.partBeginningBuffer[aDimRangeStart + 7];
			
			//------------------------------------------------------------- retrieve b and its dimensions
			int bIndex = bBuffers.partIndexBuffer[lpuIndex];
			int bStartsAt = bBuffers.partBeginningBuffer[bIndex];
			b = (double *) bBuffers.dataBuffer[bStartsAt];
			int bDimRangeStart = lpuIndex * 2 * 2 * 2; 
			bSRanges[0][0] = bBuffers.partBeginningBuffer[bDimRangeStart];
			bSRanges[0][1] = bBuffers.partBeginningBuffer[bDimRangeStart + 1];
			bSRanges[1][0] = bBuffers.partBeginningBuffer[bDimRangeStart + 2];
			bSRanges[1][1] = bBuffers.partBeginningBuffer[bDimRangeStart + 3];

			bPRanges[0][0] = bBuffers.partBeginningBuffer[bDimRangeStart + 4];
			bPRanges[0][1] = bBuffers.partBeginningBuffer[bDimRangeStart + 5];
			bPRanges[1][0] = bBuffers.partBeginningBuffer[bDimRangeStart + 6];
			bPRanges[1][1] = bBuffers.partBeginningBuffer[bDimRangeStart + 7];
			
			//------------------------------------------------------------- retrieve c and its dimensions
			int cIndex = cBuffers.partIndexBuffer[lpuIndex];
			int cStartsAt = cBuffers.partBeginningBuffer[cIndex];
			c = (double *) cBuffers.dataBuffer[cStartsAt];
			int cDimRangeStart = lpuIndex * 2 * 2 * 2; 
			cSRanges[0][0] = cBuffers.partBeginningBuffer[cDimRangeStart];
			cSRanges[0][1] = cBuffers.partBeginningBuffer[cDimRangeStart + 1];
			cSRanges[1][0] = cBuffers.partBeginningBuffer[cDimRangeStart + 2];
			cSRanges[1][1] = cBuffers.partBeginningBuffer[cDimRangeStart + 3];
			
			cPRanges[0][0] = cBuffers.partBeginningBuffer[cDimRangeStart + 4];
			cPRanges[0][1] = cBuffers.partBeginningBuffer[cDimRangeStart + 5];
			cPRanges[1][0] = cBuffers.partBeginningBuffer[cDimRangeStart + 6];
			cPRanges[1][1] = cBuffers.partBeginningBuffer[cDimRangeStart + 7];	
		}
		__syncthreads();

		/*--------------------------------------------------------------------------------------------------------------------
						Space A-Sub: Compiler Generated Space for Subpartition 
		--------------------------------------------------------------------------------------------------------------------*/

		// once we have the storage and partition dimensions of data structure at the top-level space's LPU
		// we can determine the sub-partition space's Lpu count
		int subpartitionCount = block_size_part_count(bPRanges[0], partition.blockSize);
		
		__shared__ int aPSubRanges[2][2], bPSubRanges[2][2];

		// the subpartitioned LPUs are processed one by one; remember that LPUs of sub-partitioned LPSes are
		// not supposed to be distributed
		for (int subLpu = 0; subLpu < subpartitionCount; subLpu++) {
			
			if (warpId == 0 && threadId == 0) {	
				// first we need to determine the partition dimension ranges of the two sub-
				// partitioned data structures, which are matrix A and B
				int lpuId = subLpu;
				aPSubRanges[0][0] = aPRanges[0][0];
				aPSubRanges[0][1] = aPRanges[0][1];
				block_size_part_range(aPSubRanges[1], aPRanges[1],
						subpartitionCount, lpuId, partition.blockSize, 0, 0); 
				block_size_part_range(bPSubRanges[0], bPRanges[0],
						subpartitionCount, lpuId, partition.blockSize, 0, 0); 
				bPSubRanges[1][0] = bPRanges[1][0];
				bPSubRanges[1][1] = bPRanges[1][1];
			}
			__syncthreads();

			// here we should load sub-section of A and B from the GPU card memory to the local memory
			// what about C? Or should we directly perform all computation on the card memory and rely
			// on the hardware's caching machanism to do the global and shared memory interactions?
		
			// In the multicore and segmented memory architecture cases the matrix-matrix multiplication 
			// code starts here. In the GPU, the existing partition scheme will result in only one warp 
			// within an SM doing computation for the user code. Rather the user should have the 
			// computation to be distributed to multiple warps for different smaller sub-sections of the 
			// block of matrix C using another lower level LPS

			/*------------------------------------------------------------------------------------------------------------
							   Space B: Lowest User Defined Space 
			------------------------------------------------------------------------------------------------------------*/

			// Space B LPUs will be distributed among the warps; so the parts' dimension configuration
			// should be different for different warps and we cannot have a single shared object per
			// part information as we have done in the previous LPSes. Rather, we will have a shared
			// memory pannel having 1 entry per warp to hold relevant part's dimension configuration.
			__shared__ int aSpaceBPRanges[WARP_COUNT][2][2];
			__shared__ int bSpaceBPRanges[WARP_COUNT][2][2];
			__shared__ int cSpaceBPRanges[WARP_COUNT][2][2];

			int spaceBLpuCount1 = block_size_part_count(cPRanges[0], 1);
			int spaceBLpuCount2 = block_size_part_count(cPRanges[1], partition.blockSize);
			int spaceBLpuCount = spaceBLpuCount1 * spaceBLpuCount2;

			// distribute the Space B LPUs among the warps
			for (int spaceBLpu = warpId; spaceBLpu < spaceBLpuCount; spaceBLpu += WARP_COUNT) {
				
				if (threadId == 0) {
					// construct the 2 dimensional LPU ID from the linear LPU Id
					int spaceBLpuId[2];
					spaceBLpuId[0] = spaceBLpu / spaceBLpuCount2;
					spaceBLpuId[1] = spaceBLpu % spaceBLpuCount2;
					
					//---------------------------------------------------- A part dimensions
					block_size_part_range(aSpaceBPRanges[warpId][0], aPSubRanges[0],
							spaceBLpuCount1, spaceBLpuId[0], 1, 0, 0); 
					aSpaceBPRanges[warpId][1][0] = aPSubRanges[1][0];
					aSpaceBPRanges[warpId][1][1] = aPSubRanges[1][1];

					//---------------------------------------------------- B part dimensions
					bSpaceBPRanges[warpId][0][0] = bPSubRanges[0][0];
					bSpaceBPRanges[warpId][0][1] = bPSubRanges[0][1];
					block_size_part_range(bSpaceBPRanges[warpId][1], bPSubRanges[1],
							spaceBLpuCount2, spaceBLpuId[1], 1, 0, 0); 
					
					//---------------------------------------------------- C part dimensions
					block_size_part_range(cSpaceBPRanges[warpId][0], cPRanges[0],
							spaceBLpuCount1, spaceBLpuId[0], 1, 0, 0); 
					block_size_part_range(cSpaceBPRanges[warpId][1], cPRanges[1],
							spaceBLpuCount2, spaceBLpuId[1], 1, 0, 0);
				} 
				// there is no syncthread operation needed here as updates done by a thread in a
				// warp is visible to all other threads in that warp
					
				/*----------------------------------------------------------------------------------------------------
								  Translated Computation Stage 
				----------------------------------------------------------------------------------------------------*/

				// the compute stage for IT matrix-matrix multiplication looks like the following
				// do { 
				// 	c[i][j] = c[i][j] +  a[i][k] * b[k][j]
				// } for i, j in c; k in a
				// In each warp we have 32 threads performing the same instruction in a lock-step
				// fasion. If we can make the threads working on different piece of data then we
				// can have a vectorized translation of the IT for loop without any additional data
				// synchronization among the threads. A simple static analysis of the code block 
				// should detect that i and j indices both appeared on the left hand side of the
				// enclosed statement but not the k index. So we can let different threads work on
				// different i or j values. In general, we should avoid varying both indices at the
				// same time to reduce memory bank conflicts.
				
				// But how do we select the index for distribution among threads that has the best 
				// potential for coalescing global memory and reducing shared memory accesses? The 
				// selection also need be cautious about compromising opportunities of parallelism.
				// The initial solution for this is incorporated in GPU utility library that, given,
				// a set of ranges to iterate, provides loop starting indexes and step sizes.   
				int iterableRanges[4];
				iterableRanges[0] = cSpaceBPRanges[warpId][0][0];
				iterableRanges[1] = cSpaceBPRanges[warpId][0][1];
				iterableRanges[2] = cSpaceBPRanges[warpId][1][0];
				iterableRanges[3] = cSpaceBPRanges[warpId][1][1];
				int indexesAndSteps[4];
				determineLoopIndexesAndSteps(2, threadId, iterableRanges, indexesAndSteps); 
				
				// iterate over the rows
				int iStart = indexesAndSteps[0];
				int iEnd = iterableRanges[1];
				int iStep = indexesAndSteps[1];
				for (int i = iStart; i <= iEnd; i += iStep) {

					int c_i = i - cSRanges[0][0];
					int a_i = i - aSRanges[0][0];
				
					// iterate over the columns
					int jStart = indexesAndSteps[2];
					int jEnd = iterableRanges[3];
					int jStep = indexesAndSteps[3];
					for (int j = jStart; j <= jEnd; j+= jStep) {
	
						int c_j = j - cSRanges[1][0];
						int b_j = j - bSRanges[1][0];
						
						// iterate over the common dimension
						int kStart = aSpaceBPRanges[warpId][1][0];
						int kEnd = aSpaceBPRanges[warpId][1][1];
						for (int k = kStart; k <= kEnd; k++) {
							
							int a_k = k - aSRanges[1][0];
							int b_k = k - bSRanges[0][0];

							int cIndex = c_i * (cSRanges[1][1] - cSRanges[1][0] + 1) + c_j;
							int aIndex = a_i * (aSRanges[1][1] - aSRanges[1][0] + 1) + a_k;
							int bIndex = b_k * (bSRanges[1][1] - bSRanges[1][0] + 1) + b_j;

							c[cIndex] += a[aIndex] * b[bIndex];	
						}
					}
				}
			} 
		}
	}
}


//------------------------------------------------------- MMM GPU Code Executor ------------------------------------------------------/

MMMGpuCodeExecutor::MMMGpuCodeExecutor(LpuBatchController *lpuBatchController, 
		mmm::Partition partition, 
		mmm::ArrayMetadata arrayMetadata,
		mmm::TaskGlobals *taskGlobals,
		mmm::ThreadLocals *threadLocals) 
		: GpuCodeExecutor(lpuBatchController) {

	this->partition = partition;
	this->arrayMetadata = arrayMetadata;
	this->taskGlobalsCpu = taskGlobals;
	this->taskGlobalsGpu = NULL;
	this->threadLocalsCpu = threadLocals;
	this->threadLocalsGpu = NULL;
}

void MMMGpuCodeExecutor::offloadFunction() {
	
	GpuBufferReferences aBuffers = lpuBatchController->getGpuBufferReferences("a");
	GpuBufferReferences bBuffers = lpuBatchController->getGpuBufferReferences("b");
	GpuBufferReferences cBuffers = lpuBatchController->getGpuBufferReferences("c");

	MMMLpuBatchRange batchRange;
	batchRange.lpuIdRange = currentBatchLpuRange;
	batchRange.lpuCount1 = lpuCount[0];
	batchRange.lpuCount2 = lpuCount[1];

	int threadsPerBlock = WARP_SIZE * WARP_COUNT;
	matrixMultiplyKernel<<< BLOCK_COUNT, threadsPerBlock >>>(batchRange,
			partition, arrayMetadata, 
			taskGlobalsGpu, threadLocalsGpu, 
			aBuffers, bBuffers, cBuffers);
}

void MMMGpuCodeExecutor::initialize() {

	size_t taskGlobalsSize = sizeof(taskGlobalsCpu);
	hipMalloc((void **) &taskGlobalsGpu, taskGlobalsSize);
	hipMemcpy(taskGlobalsGpu, taskGlobalsCpu, taskGlobalsSize, hipMemcpyHostToDevice);

	size_t threadLocalsSize = sizeof(threadLocalsCpu);
	hipMalloc((void **) &threadLocalsGpu, threadLocalsSize);
	hipMemcpy(threadLocalsGpu, threadLocalsCpu, threadLocalsSize, hipMemcpyHostToDevice);
}
        
void MMMGpuCodeExecutor::cleanup() {

	size_t taskGlobalsSize = sizeof(taskGlobalsCpu);
	hipMemcpy(taskGlobalsCpu, taskGlobalsGpu, taskGlobalsSize, hipMemcpyDeviceToHost);
	size_t threadLocalsSize = sizeof(threadLocalsCpu);
	hipMemcpy(threadLocalsCpu, threadLocalsGpu, threadLocalsSize, hipMemcpyDeviceToHost);

	GpuCodeExecutor::cleanup();
}

