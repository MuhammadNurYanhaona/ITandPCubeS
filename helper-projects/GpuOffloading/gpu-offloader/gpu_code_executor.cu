#include "gpu_code_executor.h"
#include "lpu_parts_tracking.h"
#include "../runtime/structure.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

//--------------------------------------------------------- GPU Code Executor -------------------------------------------------------------/

GpuCodeExecutor::GpuCodeExecutor(LpuBatchController *lpuBatchController) {
	this->lpuBatchController = lpuBatchController;
}

void GpuCodeExecutor::submitNextLpu(LPU *lpu) {
	
	if (lpuBatchController->canAddNewLpu() && lpuBatchController->canHoldLpu(lpu)) {
		lpuBatchController->addLpuToTheCurrentBatch(lpu);
		return;	
	}
	if (!lpuBatchController->isEmptyBatch()) {
		execute();
	}
	lpuBatchController->addLpuToTheCurrentBatch(lpu);
}

void GpuCodeExecutor::forceExecution() {
	if (!lpuBatchController->isEmptyBatch()) {
		execute();
	}
}

void GpuCodeExecutor::execute() {
	lpuBatchController->submitCurrentBatchToGpu();
	offloadFunction();
	hipDeviceSynchronize();
	lpuBatchController->updateBatchDataPartsFromGpuResults();
	lpuBatchController->resetController();
}
